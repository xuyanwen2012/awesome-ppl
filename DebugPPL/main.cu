﻿#include <iostream>
#include <random>
#include <memory>
#include <glm/glm.hpp>

#include "cuda/dispatcher.h"
#include "cuda/helper.cuh"
#include "shared/structures.h"
#include "shared/brt_func.h" // this is actually a CPU function

#include <gtest/gtest.h>

#include <execution> // just for speeding up the test

// ------------------ configs ------------------
//constexpr auto n = 1920 * 1080; // ~2M

constexpr auto n = 640 * 480; // ~300k
constexpr auto min_coord = 0.0f;
constexpr auto range = 1024.0f;
constexpr auto seed = 114514;

[[nodiscard]] std::unique_ptr<pipe> generate_pipe()
{
	auto gpu_pip = std::make_unique<pipe>(n, min_coord, range, seed);

	std::mt19937 gen(seed); // NOLINT(cert-msc51-cpp)
	std::uniform_real_distribution dis(min_coord, min_coord + range);
	std::generate_n(std::execution::seq, gpu_pip->u_points, n, [&dis, &gen]
	{
		return glm::vec4(dis(gen), dis(gen), dis(gen), 1.0f);
	});

	return gpu_pip;
}

// ===============================================
//	Morton + Sort
// ===============================================

void test_morton_and_sort(const int grid_size)
{
	auto gpu_pip = generate_pipe();
	const auto cpu_points = std::vector(gpu_pip->u_points, gpu_pip->u_points + n);

	// ------- testing region ------------
	constexpr auto stream_id = 0;
	gpu::dispatch_ComputeMorton(grid_size, stream_id, *gpu_pip);
	gpu::dispatch_RadixSort(grid_size, stream_id, *gpu_pip);
	gpu::sync_stream(stream_id);
	// -----------------------------------

	// generate CPU result
	std::vector<morton_t> cpu_morton(n);
	std::transform(std::execution::par, cpu_points.begin(), cpu_points.end(), cpu_morton.begin(), [&](const auto& p)
	{
		return shared::xyz_to_morton32(p, min_coord, range);
	});
	std::sort(std::execution::par, cpu_morton.begin(), cpu_morton.end());

	const auto is_sorted = std::is_sorted(gpu_pip->u_morton, gpu_pip->u_morton + n);
	EXPECT_TRUE(is_sorted);

	const auto is_equal = std::equal(cpu_morton.begin(), cpu_morton.end(), gpu_pip->u_morton);
	EXPECT_TRUE(is_equal);
}


TEST(ComputeMorton, GridSize)
{
	for (auto i = 1; i < 16; i++)
	{
		EXPECT_NO_FATAL_FAILURE(test_morton_and_sort(i));
	}
}

// ===============================================
//	Unique
// ===============================================

void test_unique(const int grid_size)
{
	auto gpu_pip = generate_pipe();

	// generate GPU result
	constexpr auto stream_id = 0;
	gpu::dispatch_ComputeMorton(grid_size, stream_id, *gpu_pip);
	gpu::dispatch_RadixSort(grid_size, stream_id, *gpu_pip);
	SYNC_DEVICE();

	// generate CPU result (assume previous test is correct)
	const std::vector cpu_morton(gpu_pip->u_morton, gpu_pip->u_morton + n);

	// ------- testing region ------------
	gpu::dispatch_RemoveDuplicates_sync(grid_size, stream_id, *gpu_pip);
	gpu::sync_stream(stream_id);
	const auto gpu_n_unique = gpu_pip->n_unique_mortons();
	// -----------------------------------

	std::vector<morton_t> cpu_morton_alt(n);
	const auto last = std::unique_copy(cpu_morton.begin(), cpu_morton.end(), cpu_morton_alt.begin());
	const auto cpu_n_unique = std::distance(cpu_morton_alt.begin(), last);

	EXPECT_EQ(cpu_n_unique, gpu_n_unique);

	// the data after 'n_unique' is 0 in GPU version
	for (auto i = 0; i < gpu_n_unique; i++)
	{
		EXPECT_EQ(cpu_morton_alt[i], gpu_pip->u_morton_alt[i]);
	}
}

TEST(Unique, GridSize)
{
	for (auto i = 1; i < 16; i++)
	{
		EXPECT_NO_FATAL_FAILURE(test_unique(i));
	}
}

// ===============================================
//	Binary Radix Tree
// ===============================================

void test_binary_radix_tree(const int grid_size)
{
	auto gpu_pip = generate_pipe();

	// generate GPU result
	constexpr auto stream_id = 0;
	gpu::dispatch_ComputeMorton(grid_size, stream_id, *gpu_pip);
	gpu::dispatch_RadixSort(grid_size, stream_id, *gpu_pip);
	gpu::dispatch_RemoveDuplicates_sync(grid_size, stream_id, *gpu_pip);
	SYNC_DEVICE();

	// ------- testing region ------------
	gpu::dispatch_BuildRadixTree(grid_size, stream_id, *gpu_pip);
	gpu::sync_stream(stream_id);
	// -----------------------------------

	auto cpu_pip = generate_pipe();
	gpu::dispatch_ComputeMorton(grid_size, stream_id, *cpu_pip);
	gpu::dispatch_RadixSort(grid_size, stream_id, *cpu_pip);
	gpu::dispatch_RemoveDuplicates_sync(grid_size, stream_id, *cpu_pip);
	SYNC_DEVICE();

	for (auto i = 0; i < cpu_pip->n_unique_mortons(); i++)
	{
		cpu::process_radix_tree_i(i, cpu_pip->n_unique_mortons(), cpu_pip->getSortedKeys(), &cpu_pip->brt);
	}

	EXPECT_EQ(cpu_pip->n_brt_nodes(), gpu_pip->n_brt_nodes());

	for (auto i = 0; i < cpu_pip->n_brt_nodes() / 2; i++)
	{
		EXPECT_EQ(cpu_pip->brt.u_prefix_n[i],
		          gpu_pip->brt.u_prefix_n[i]);
	}
}

TEST(BinaryRadixTree, GridSize)
{
	EXPECT_NO_FATAL_FAILURE(test_binary_radix_tree(16));
}

int main(int argc, char** argv)
{
	// some setups
	constexpr auto n_streams = 1;
	gpu::initialize_dispatcher(n_streams);

	testing::InitGoogleTest(&argc, argv);
	const auto ret = RUN_ALL_TESTS();

	gpu::release_dispatcher();
	return ret;
}
